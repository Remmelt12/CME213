#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>

#define BLOCK_SIZE 16


__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    return result;
}

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/

__global__
void myGEMMkernel(double* A, double* B, double* C, double alpha, double beta, int M,
           int N, int K,bool AT,bool BT) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double inner_prod=0.0;
    if(row<M&&col<N){
        for(int k = 0; k<K; k++){
            int indexA;
            int indexB;
            if (AT){
                 indexA = (row*K)+k;
                 indexB = (col*K)+k;
            
            }

            else if(BT){
                indexA = (k*M)+row;
                indexB = (k*N)+col;
            
            }

            else{
                indexA = (k*M)+row;
                indexB = (col*K)+k;
            
            }
            inner_prod+=A[indexA]*B[indexB];
        }
        C[col*M+row] =alpha*inner_prod+beta*C[col*M+row];
    }
}

int myGEMM(double* A, double* B, double* C, double* alpha, double* beta, int M,
           int N, int K,bool AT,bool BT) 
{
    /* TODO: Write an efficient GEMM implementation on GPU */
    dim3 dimBlock(32,32);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x, (M+dimBlock.y-1)/dimBlock.y);

    myGEMMkernel<<<dimGrid, dimBlock>>>(A,B,C,*alpha,*beta,M,N,K,AT,BT);

    return 0;
}

__global__
void softmax_kernel(const double* Z, double* A,int M, int N)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double denom = 0.0;
    for (int i =0;i<M;i++)
    {
        denom+=exp(Z[col*M+i]);
    }
    for (int i =0;i<M;i++)
    {
        A[col*M+i]=exp(Z[col*M+i])/denom;
    }
}

void softmax_p(const double* Z, double* A,int M, int N)
{
    dim3 dimBlock(32);
    dim3 dimGrid((N+dimBlock.y-1)/dimBlock.x);
    softmax_kernel<<<dimGrid,dimBlock>>>(Z,A,M,N);

}

__global__
void sigmoid_kernel(const double* Z, double* A,int M,int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row<M&&col<N)
    {
        A[M*col+row]=1.0/(1.0+exp(-1*Z[M*col+row]));
    }
}

void sigmoid_p(const double* Z, double* A,int M, int N)
{
    dim3 dimBlock(32,32);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x, (M+dimBlock.y-1)/dimBlock.y);
    sigmoid_kernel<<<dimGrid,dimBlock>>>(Z,A,M,N);
}

__global__
void row_sum_kernel(double* W, double* Y, int M, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M)
    {
        double sum=0.0;
        for(int i=0;i<N;i++){
            sum+=W[i*M+row];
        }
        Y[row]=sum;
    }
    
}


void row_sum(double* W, double* Y, int M, int N)
{
    dim3 dimBlock(32);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);
    row_sum_kernel<<<dimGrid,dimBlock>>>(W,Y,M,N);


}
__global__
void elem_mult_kernel(double* A, double* B, double* C,int M,int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row<M&&col<N)
    {
        C[(col*M+row)]=A[(col*M+row)]*B[(col*M+row)];
    }

}
void elem_mult(double* A, double* B, double* C,int M,int N)
{
    dim3 dimBlock(32,32);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x, (M+dimBlock.y-1)/dimBlock.y);
    elem_mult_kernel<<<dimGrid,dimBlock>>>(A,B,C,M,N);
}
