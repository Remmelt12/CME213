#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>

#define BLOCK_SIZE 16


__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    return result;
}

_global__
void myGEMM_kernel(double* A, double* B, double* C,
                   double alpha, double beta,
                   int M, int N, int K,
                   bool AT, bool BT) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int c_ind = row + (col * M);
        double dot_prod = 0.0;
        int a_ind;
        int b_ind;
        for(int i = 0; i < K; i++) {
            if (AT)
                a_ind = (row*K) + i;
            else
                a_ind = row + (i*M);
            if (BT)
                b_ind = col + (i*N);
            else
                b_ind = i + (col * K);
            dot_prod += A[a_ind] * B[b_ind];
        }
        C[c_ind] = (alpha * dot_prod) + (beta * C[c_ind]);
    }
}

/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/
int myGEMM(double* A, double* B, double* C,
           double* alpha, double* beta,
           int M, int N, int K,
           bool AT, bool BT) {
    /* TODO: Write an efficient GEMM implementation on GPU */
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    myGEMM_kernel<<< blocks, threads >>>(A, B, C, *alpha, *beta, M, N, K, AT, BT);
    check_launch("myGEMM_kernel");
    return 0;
}

/* GPU kernel for 10-class softmax */
__global__
void gpuSoftmax_kernel(double* A, unsigned int num_classes, unsigned int N) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    if (col < N) {
        double denominator = 0.0;

        for(int c = 0; c < num_classes; c++){
            denominator += (double) std::exp(A[col*num_classes + c]);
        }

        for(int c = 0; c < num_classes; c++){
            int ij = c + (col * num_classes);
            A[ij] = (double) std::exp(A[ij])/ (double) denominator;
        }
    }
}

/* Routine for 10-class softmax */
void gpuSoftmax(double* A, unsigned int num_classes, unsigned int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = num_threads;
    dim3 threads(thr_x);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    dim3 blocks(blk_x);

    gpuSoftmax_kernel<<< blocks, threads >>>(A, num_classes, N);
    check_launch("gpuSoftmax_kernel");
}

/* GPU kernel for in-place element-wise sigmoid */
__global__
void gpuSigmoid_kernel(double* A, unsigned int num_neurons, unsigned int N) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(col < N && row < num_neurons) {
        int ij = row + (col * num_neurons);
        A[ij] = (double) 1.0 / (double)(1.0 + exp(-1.0 * A[ij]));
    }
}

/* Routine for in-place element-wise sigmoid */
void gpuSigmoid(double* A, unsigned int num_neurons, unsigned int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (num_neurons + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuSigmoid_kernel<<< blocks, threads >>>(A, num_neurons, N);
    check_launch("gpuSigmoid_kernel");
}

/* GPU kernel for summing rows of matrix A. Places row sums in vector v*/
__global__
void gpuRowSum_kernel(double *A, double *v, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M) {
        double rowSum = 0.0;
        for (int i = 0; i < N; i++) 
            rowSum += A[(M*i) + row];
        v[row] = rowSum;
    }
}

/* Routine for summing rows of matrix A. Places row sums in vector v */
void gpuRowSum(double *A, double *v, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = num_threads;
    dim3 threads(thr_x);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    dim3 blocks(blk_x);

    gpuRowSum_kernel<<< blocks, threads >>>(A, v, M, N);
    check_launch("gpuRowSum_kernel");
}

/* GPU kernel for broadcasting sum for matrix A with vector v */
__global__
void gpuMatVecSum_kernel(double *A, double *v, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; 
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M &&  col < N) {
        int ind = row + (M*col);
        double num = v[row];
        A[ind] += num;
    }
}

/* Routine for broadcasting sum for matrix A with vector v */
void gpuMatVecSum(double *A, double *v, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);
    gpuMatVecSum_kernel<<< blocks, threads >>>(A, v, M, N);
    check_launch("gpuMatVecSum_kernel");
}

/* GPU kernel for elementwise matrix sum */
__global__
void gpuElementwiseSum_kernel(double *A, double *B, double *C,
                              double alpha, double beta,
                              int M, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < M && col < N) {
        int ind = row + (M*col);
        C[ind] = (alpha * A[ind]) + (beta * B[ind]);
    }
}

/* Routine for elementwise matrix sum */
void gpuElementwiseSum(double *A, double *B, double *C, 
                       double alpha, double beta,
                       int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (N + thr_x - 1) / thr_x;
    unsigned int blk_y = (M + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpuElementwiseSum_kernel<<< blocks, threads >>>(A, B, C, alpha, beta, M, N);
    check_launch("gpuElementwiseSum_kernel");
}

/* GPU kernel for derivative of sigmoid */
__global__
void gpudSigmoid_kernel(double *A, double *B, double *C, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < M && col < N) {
        int ind = row + (M*col);
        C[ind] = (double) A[ind] * B[ind] * (1.0 - B[ind]);
    }
}

/** Routine for derivative of sigmoid */
void gpudSigmoid(double *A, double *B, double *C, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    gpudSigmoid_kernel<<< blocks, threads >>>(A, B, C, M, N);
    check_launch("gpudSigmoid");
}
/*
Routine to perform an in-place GEMM operation, i.e., C := alpha*A*B + beta*C
*/

/*
__global__
void myGEMMkernel(double* A, double* B, double* C, double alpha, double beta, int M,
           int N, int K,bool AT,bool BT) 
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < M && col < N) {
        int c_ind = row + (col * M);
        double dot_prod = 0.0;
        int a_ind;
        int b_ind;
        for(int i = 0; i < K; i++) {
            if (AT)
                a_ind = (row*K) + i;
            else
                a_ind = row + (i*M);
            if (BT)
                b_ind = col + (i*N);
            else
                b_ind = i + (col * K);
            dot_prod += A[a_ind] * B[b_ind];
        }
        C[c_ind] = (alpha * dot_prod) + (beta * C[c_ind]);
    }
    /*
    double inner_prod=0.0;
            if (AT){
			if(row< K&&col <N){
				for(int k = 0; k<K; k++){
                int indexA = (row*K)+k;
                int indexB = (col*K)+k;
				inner_prod+=A[indexA]*B[indexB];
            
				}
            }
			}
            else if(BT){
			if(row< M&&col <K){
				for(int k = 0; k<K; k++){
                int indexA = (k*M)+row;
                int indexB = (k*N)+col;
				inner_prod+=A[indexA]*B[indexB];
            }
			}
			}

            else{
			if(row< M&&col <N){
				for(int k = 0; k<K; k++){
					int indexA = (k*M)+row;
					int indexB = (col*K)+k;
					inner_prod+=A[indexA]*B[indexB];
            
            }
        }
    }
        C[col*M+row] =alpha*inner_prod+beta*C[col*M+row];
      */  

}

int myGEMM(double* A, double* B, double* C, double* alpha, double* beta, int M,
           int N, int K,bool AT,bool BT) 
{
    /* TODO: Write an efficient GEMM implementation on GPU */
    dim3 dimBlock(32,6);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);

    myGEMMkernel<<<dimGrid, dimBlock>>>(A,B,C,*alpha,*beta,M,N,K,AT,BT);

    return 0;
}

__global__
void softmax_kernel(double* A,int M, int N)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < N) {
        double denominator = 0.0;

        for(int c = 0; c < M; c++){
            denominator += (double) std::exp(A[col*M + c]);
        }

        for(int c = 0; c < M; c++){
            int ij = c + (col * M);
            A[ij] = (double) std::exp(A[ij])/ (double) denominator;
        }
    }
    /*
    double denom = 0.0;
    if(col<N){
    for (int i =0;i<M;i++)
    {
        denom+=(double) std::exp(Z[col*M+i]);
    }
    for (int i =0;i<M;i++)
    {
        A[col*M+i]=std::exp(Z[col*M+i])/(double)denom;
    }
    }*/
}

void softmax_p(double* A,int M, int N)
{
    dim3 dimBlock(192);
    dim3 dimGrid((N+dimBlock.y-1)/dimBlock.x);
    softmax_kernel<<<dimGrid,dimBlock>>>(A,M,N);

}

__global__
void sigmoid_kernel(double* A,int M,int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M&&col<N)
    {
        int ind = row + (col * M);
        A[ind] = (double) 1.0 / (double)(1.0 + exp(-1.0 * A[ind]));
    }
}

void sigmoid_p(double* A,int M, int N)
{
    dim3 dimBlock(32,6);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    sigmoid_kernel<<<dimGrid,dimBlock>>>(A,M,N);
}

__global__
void row_sum_kernel(double* W, double* Y, int M, int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row<M)
    {
        double sum=0.0;
        for(int i=0;i<N;i++){
            sum+=W[(i*M)+row];
        }
        Y[row]=sum;
    }
    
}


void row_sum(double* W, double* Y, int M, int N)
{
    dim3 dimBlock(192);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x);
    row_sum_kernel<<<dimGrid,dimBlock>>>(W,Y,M,N);


}

__global__
void elem_mult_kernel(double* A, double* B, double* C,int M,int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M&&col<N)
    {
        C[(col*M+row)]=A[(col*M+row)]*B[(col*M+row)];
    }

}
void elem_mult(double* A, double* B, double* C,int M,int N)
{
    dim3 dimBlock(32,32);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    elem_mult_kernel<<<dimGrid,dimBlock>>>(A,B,C,M,N);
}

__global__
void elem_add_kernel(double* A, double* B, double* C,double alpha, double beta,int M,int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M&&col<N)
    {
        C[(col*M+row)]=alpha*A[(col*M+row)]+beta*B[(col*M+row)];
    }

}

void elem_add(double* A, double* B, double* C,double alpha, double beta,int M,int N)
{
    dim3 dimBlock(32,32);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    elem_add_kernel<<<dimGrid,dimBlock>>>(A,B,C,alpha,beta,M,N);
}

__global__
void elem_mod_kernel(double* A, double* B,double alpha, double beta,int M,int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M&&col<N)
    {
        B[(col*M+row)]=alpha-beta*A[(col*M+row)];
    }

}

void elem_mod(double* A, double* B,double alpha, double beta,int M,int N)
{
    dim3 dimBlock(32,32);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    elem_mod_kernel<<<dimGrid,dimBlock>>>(A,B,alpha,beta,M,N);



}
/* GPU kernel for derivative of sigmoid */

__global__
void sigmoid_back_kernel(double *A, double *B, double *C, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N) {
        C[(M*col)+row]=A[(M*col)+row] * B[(M*col)+row] * (1.0 -B[(M*col)+row]);
    }

}

/** Routine for derivative of sigmoid */
void sigmoid_back(double *A, double *B, double *C, int M, int N) {
    dim3 dimBlock(32,6);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    sigmoid_back_kernel<<< dimGrid, dimBlock >>>(A, B, C, M, N);
}
*/
