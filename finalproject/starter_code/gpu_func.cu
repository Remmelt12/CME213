#include "hip/hip_runtime.h"
#include "gpu_func.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include "hipblas.h"
#include <cmath>

#define BLOCK_SIZE 32
__global__
void shared_GEMM_kernel(double* __restrict__ A, double* __restrict__ B, double* __restrict__ C,
                  double alpha, double beta,
                  int M, int N, int K) {
    int by = blockIdx.y;
    int bx = blockIdx.x;
    int ty = threadIdx.y;
    int tx = threadIdx.x;
    __shared__ double As[BLOCK_SIZE][BLOCK_SIZE + 1];
    __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE + 1];

    double C_aggr = 0.0;
    int lim = (K + BLOCK_SIZE - 1)/BLOCK_SIZE;
    for (int i = 0; i < lim; ++i) {
        //fill in shared memory
        if ((BLOCK_SIZE * i) < (K - ty)) {
            double *A_part = A + (M * BLOCK_SIZE * i + BLOCK_SIZE * bx);
            int ia = (ty * M) + tx;
            As[tx][ty] = A_part[ia];
        }
        else {
            As[tx][ty] = 0.0;
        }

        if ((BLOCK_SIZE * i) < (K - tx)) {
            double *B_part = B + (K * BLOCK_SIZE * by + BLOCK_SIZE * i);
            int ib = (ty * K) +  tx;
            Bs[tx][ty] = B_part[ib];
        }
        else{
            Bs[tx][ty] = 0.0;
        }

        __syncthreads();

        //matrix multiplication
#pragma unroll
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            C_aggr += As[tx][j] * Bs[j][ty];
        }

        __syncthreads();
    }


    int col = BLOCK_SIZE * by + ty;
    int row = BLOCK_SIZE * bx + tx;
    if((col < N) && (row < M)){
        int ic = col * M + row;
        C[ic] =alpha*C_aggr+ beta*C[ic];
    }
}

__global__
void shared_GEMM_kernel1(double* __restrict__ A, double* __restrict__ B, double* __restrict__ C,
                     double alpha, double beta,
                     int M, int N, int K) {
    int bx = blockIdx.y;
    int by = blockIdx.x;
    int tx = threadIdx.y;
    int ty = threadIdx.x;

    __shared__ double As[BLOCK_SIZE][BLOCK_SIZE+1];
    __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE+1];

    double C_aggr = 0;
    int lim = (K + BLOCK_SIZE - 1) /  BLOCK_SIZE;
    for (int i = 0; i < lim; ++i) {
        // fill in shared memory
        if(BLOCK_SIZE * i < K - ty) {
            double* A_part = A + (K * BLOCK_SIZE * bx + BLOCK_SIZE * i);
            int ia = (tx * K) + ty;
            As[tx][ty] = A_part[ia];
        }
        else
            As[tx][ty] = 0;    

        if(BLOCK_SIZE * i < K - tx) {
            double* B_part = B + (K * BLOCK_SIZE * by + BLOCK_SIZE * i);
            int ib = (ty * K) + tx;
            Bs[tx][ty] = B_part[ib];
        }
        else
            Bs[tx][ty] = 0;

        __syncthreads();
        
        //matrix multiplication
#pragma unroll
        for (int j = 0; j < BLOCK_SIZE; ++j)
            C_aggr += As[tx][j] * Bs[j][ty];
        __syncthreads();
    }


    int col = BLOCK_SIZE * by + ty;
    int row = BLOCK_SIZE * bx + tx;
    if((col < N) && (row < M)){
        int ic = col * M + row;
        C[ic] =alpha*C_aggr+ beta*C[ic];
    }
}
__global__
void shared_GEMM_kernel2(double* __restrict__ A, double* __restrict__ B, double* __restrict__ C,
                     double alpha, double beta,
                     int M, int N, int K) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ double As[BLOCK_SIZE][BLOCK_SIZE+1];
    __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE+1];

    double C_aggr = 0;
    int lim = (K + BLOCK_SIZE - 1) /  BLOCK_SIZE;

    for (int i = 0; i < lim; ++i) {
        // fill in shared memory
        if(BLOCK_SIZE * i < K - ty) {
            double* A_part = A + (M * BLOCK_SIZE * i + BLOCK_SIZE * bx);
            int ia = (ty * M) + tx;
            As[tx][ty] = A_part[ia];
        }
        else
            As[tx][ty] = 0;    

        if(BLOCK_SIZE * i < K - tx) {
            double* B_part = B + (N * BLOCK_SIZE * i + BLOCK_SIZE * by);
            int ib = (tx * N) + ty;
            Bs[tx][ty] = B_part[ib];
        }
        else
            Bs[tx][ty] = 0;

        __syncthreads();
        
        //matrix multiplication
#pragma unroll
        for (int j = 0; j < BLOCK_SIZE; ++j)
            C_aggr += As[tx][j] * Bs[j][ty];
        __syncthreads();
    }


    int col = BLOCK_SIZE * by + ty;
    int row = BLOCK_SIZE * bx + tx;
    if((col < N) && (row < M)){
        int ic = col * M + row;
        C[ic] =alpha*C_aggr+ beta*C[ic];
    }
}

__global__
void device_add_one(int* d_result, int t) {
    *d_result = t + 1;
}

/*
Just a dummy function that can be used to warm up GPU
*/
int useless_gpu_add_one(int t) {
    int result;
    int* d_result;

    checkCudaErrors(hipMalloc((void**)&d_result, 1 * sizeof(int)));

    event_pair timer;
    start_timer(&timer);
    device_add_one<<<1,1>>>(d_result, t);
    check_launch("device_add_one");
    double time = stop_timer(&timer);

    std::cout << "device_add_one took: " << time << " seconds" << std::endl;

    checkCudaErrors(hipMemcpy(&result, d_result, 1 * sizeof(int),
                               hipMemcpyDeviceToHost));
    return result;
}

__global__
void myGEMMkernel(double* A, double* B, double* C, double alpha, double beta, int M,
           int N, int K) 
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(row<M && col<N )
    {
        double inner_prod=0.0;
        int a_ind;
        int b_ind;
        for(int i = 0; i < K; i++) {
            a_ind = row + (i*M);
            b_ind = i + (col * K);
            inner_prod += A[a_ind] * B[b_ind];
        }
    
        C[col*M+row] =alpha*inner_prod+beta*C[col*M+row];
    }
}

__global__
void myGEMMkernel1(double* A, double* B, double* C, double alpha, double beta, int M,
           int N, int K) 
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M && col< N)
    {
        int ic=(col*M)+row;
        double inner_prod=0.0;
        int ia;
        int ib;
        for(int i=0;i<K;i++)
        {
            ia=(row*K)+i;
            ib=i+(col*K);
            inner_prod+= A[ia]*B[ib];
        }
        C[ic] =alpha*inner_prod+beta*C[ic];
        
    }

}

__global__
void myGEMMkernel2(double* A, double* B, double* C, double alpha, double beta, int M,
           int N, int K) 
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M && col< N)
    {
        int ic=(col*M)+row;
        double inner_prod=0.0;
        int ia;
        int ib;
        for(int i=0;i<K;i++)
        {
            ia=(i*M)+row;
            ib=col+(i*N);
            inner_prod+= A[ia]*B[ib];
        }
        C[ic] =alpha*inner_prod+beta*C[ic];
        
    }

}
int myGEMM(double* A, double* B, double* C, double* alpha, double* beta, int M,
           int N, int K,bool AT,bool BT) 
{
   if (AT){
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        int blk_x = (N+dimBlock.x-1)/  dimBlock.x;
        int blk_y = (M+dimBlock.x-1) / dimBlock.y;
        dim3 dimGrid(blk_x, blk_y);

        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

        shared_GEMM_kernel1<<<dimGrid, dimBlock>>>(A, B, C, *alpha, *beta, M, N, K);
    }
    else if (BT) {
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        int blk_x = (M + dimBlock.x - 1)/dimBlock.x;
        int blk_y = (N + dimBlock.y - 1)/dimBlock.y;
        dim3 dimGrid (blk_x, blk_y);

        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

        shared_GEMM_kernel2 <<<dimGrid, dimBlock>>> (A, B, C, *alpha, *beta, M, N, K);
    }
    else{
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        int blk_x = (M + dimBlock.x - 1)/dimBlock.x;
        int blk_y = (N + dimBlock.y - 1)/dimBlock.y;
        dim3 dimGrid (blk_x, blk_y);

        hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);

        shared_GEMM_kernel <<<dimGrid, dimBlock>>> (A, B, C, *alpha, *beta, M, N, K);
    }
    return 0;
}

/* GPU kernel for 10-class softmax */
__global__
void gpuSoftmax_kernel(double* A, unsigned int num_classes, unsigned int N) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    if (col < N) {
        double denominator = 0.0;

        for(int c = 0; c < num_classes; c++){
            denominator += (double) std::exp(A[col*num_classes + c]);
        }

        for(int c = 0; c < num_classes; c++){
            int ij = c + (col * num_classes);
            A[ij] = (double) std::exp(A[ij])/ (double) denominator;
        }
    }
}

__global__
void softmax_kernel(double* A,int M, int N)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double denom = 0.0;
    if(col<N){
    for (int i =0;i<M;i++)
    {
        denom+=(double) std::exp(A[col*M+i]);
    }
    for (int i =0;i<M;i++)
    {
        A[col*M+i]=std::exp(A[col*M+i])/(double)denom;
    }
    }
}

__global__
void sigmoid_kernel(double* A,int M,int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M&&col<N)
    {
        int ind = row + (col * M);
        A[ind] = (double) 1.0 / (double)(1.0 + exp(-1.0 * A[ind]));
    }
}

/* Routine for in-place element-wise sigmoid */
void gpuSigmoid(double* A, unsigned int num_neurons, unsigned int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (num_neurons + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);

    sigmoid_kernel<<< blocks, threads >>>(A, num_neurons, N);
    check_launch("gpuSigmoid_kernel");
}


__global__
void row_sum_kernel(double* W, double* Y, int M, int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row<M)
    {
        double sum=0.0;
        for(int i=0;i<N;i++){
            sum+=W[(i*M)+row];
        }
        Y[row]=sum;
    }
    
}

/* GPU kernel for broadcasting sum for matrix A with vector v */
__global__
void gpuMatVecSum_kernel(double *A, double *v, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; 
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M &&  col < N) {
        int ind = row + (M*col);
        double num = v[row];
        A[ind] += num;
    }
}

/* Routine for broadcasting sum for matrix A with vector v */
void gpuMatVecSum(double *A, double *v, int M, int N) {
    unsigned int num_threads = 192;
    unsigned int thr_x = 32;
    unsigned int thr_y = (num_threads + thr_x - 1) / thr_x;
    dim3 threads(thr_x, thr_y);

    unsigned int blk_x = (M + thr_x - 1) / thr_x;
    unsigned int blk_y = (N + thr_y - 1) / thr_y;
    dim3 blocks(blk_x, blk_y);
    gpuMatVecSum_kernel<<< blocks, threads >>>(A, v, M, N);
    check_launch("gpuMatVecSum_kernel");
}

/* GPU kernel for elementwise matrix sum */

__global__
void elem_add_kernel(double* A, double* B, double* C,double alpha, double beta,int M,int N)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row<M&&col<N)
    {
        int index =(M*col)+row;
        C[index]=(alpha*A[index])+(beta*B[index]);
    }

}


/* GPU kernel for derivative of sigmoid */
/** Routine for derivative of sigmoid */

__global__
void sigmoid_back_kernel(double *A, double *B, double *C, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < M && col < N) {
        int index =(M*col)+row;
        C[index]=(double)A[index] * B[index] * (1.0 -B[index]);
    }

}


void softmax_p(double* A,int M, int N)
{
    dim3 dimBlock(32);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);
    softmax_kernel<<<dimGrid,dimBlock>>>(A,M,N);

}


void sigmoid_p(double* A,int M, int N)
{
    dim3 dimBlock(32,6);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    sigmoid_kernel<<<dimGrid,dimBlock>>>(A,M,N);
}



void row_sum(double* W, double* Y, int M, int N)
{
    dim3 dimBlock(192);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x);
    row_sum_kernel<<<dimGrid,dimBlock>>>(W,Y,M,N);


}


void elem_add(double* A, double* B, double* C,double alpha, double beta,int M,int N)
{
    dim3 dimBlock(32,32);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    elem_add_kernel<<<dimGrid,dimBlock>>>(A,B,C,alpha,beta,M,N);
}

void sigmoid_back(double *A, double *B, double *C, int M, int N) {
    dim3 dimBlock(32,6);
    dim3 dimGrid((M+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    sigmoid_back_kernel<<< dimGrid, dimBlock >>>(A, B, C, M, N);
}

