#include "hip/hip_runtime.h"
#include <algorithm>
#include <cctype>
#include <fstream>
#include <iostream>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <thrust/binary_search.h>
#include <thrust/adjacent_difference.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

// You may include other thrust headers if necessary.

#include "test_macros.h"

// You will need to call these functors from
// thrust functions in the code do not create new ones

// returns true if the char is not a lowercase letter
struct isnot_lowercase_alpha : thrust::unary_function<unsigned char, bool> {
   __host__ __device__ 
   bool operator()(unsigned char c){return(c<'a' || c>'z');} 
};

// convert an uppercase letter into a lowercase one
// do not use the builtin C function or anything from boost, etc.
struct upper_to_lower : thrust::unary_function<unsigned char, unsigned char> {

    unsigned char operator()(unsigned char c)
    {
        if(check_(c)){
            return c+32;
        }
        else{
            return c;
        }
    }

    private:
    isnot_lowercase_alpha check_;
};

// apply a shift with appropriate wrapping
struct apply_shift : thrust::binary_function<unsigned char, int,
        unsigned char> {
    unsigned char operator()(unsigned char c, int position)
    {
        int key_pos=position % period_;
        char shift = begin_[key_pos]-97;
        return c+shift ; 
    }

    apply_shift(char* begin,unsigned int period): begin_(begin), period_(period) {}
    
    private:
        char* begin_;
        unsigned int period_;

};

// Returns a vector with the top 5 letter frequencies in text.
std::vector<double> getLetterFrequencyCpu(
    const std::vector<unsigned char>& text) {
    std::vector<unsigned int> freq(256);

    for(unsigned int i = 0; i < text.size(); ++i) {
        freq[tolower(text[i])]++;
    }

    unsigned int sum_chars = 0;

    for(unsigned char c = 'a'; c <= 'z'; ++c) {
        sum_chars += freq[c];
    }

    std::vector<double> freq_alpha_lower;

    for(unsigned char c = 'a'; c <= 'z'; ++c) {
        if(freq[c] > 0) {
            freq_alpha_lower.push_back(freq[c] / static_cast<double>(sum_chars));
        }
    }

    std::sort(freq_alpha_lower.begin(), freq_alpha_lower.end(),
              std::greater<double>());
    freq_alpha_lower.resize(min(static_cast<int>(freq_alpha_lower.size()), 5));

    return freq_alpha_lower;
}

// Print the top 5 letter frequencies and them.
std::vector<double> getLetterFrequencyGpu(
    const thrust::device_vector<unsigned char>& text) {

    std::vector<double> freq_alpha_lower;
    
    unsigned int sum_chars = 0;

    // WARNING: make sure you handle the case of not all letters appearing
    // in the text.


    // TODO calculate letter frequency
    // copy input data 
    thrust::device_vector<unsigned char> data(text);
    thrust::device_vector<unsigned int> freq;

    // sort data to bring equal elements together
    thrust::sort(data.begin(), data.end());

    // number of freq bins is equal to the maximum
    // value plus one
    unsigned int num_bins = data.back() + 1;

    // resize freq storage
    freq.resize(num_bins);
    freq_alpha_lower.resize(num_bins);

    // find the end of each bin of values
    thrust::counting_iterator<unsigned int> search_begin(0);
    thrust::upper_bound(data.begin(),data.end(),
                        search_begin,search_begin + num_bins,
                        freq.begin());

    // compute the freq by taking
    // differences of the cumulative
    // freq
    thrust::adjacent_difference(freq.begin(),
                                freq.end(),
                                freq.begin());

    // print the freq
    sum_chars=thrust::reduce(freq.begin(),freq.end());

    for(unsigned char c = 'a'; c <= 'z'; ++c) {
        if(freq[c] > 0) {
            freq_alpha_lower.push_back(freq[c] / static_cast<double>(sum_chars));
        }
    }

    /*
    thrust::transform(freq.begin(),freq.end(),freq_alpha_lower,[=]__host__
            __device__ (unsigned int i) double {return ((double) i)/((double)
                sum_chars) });
    */
    thrust::sort(freq_alpha_lower.begin(),freq_alpha_lower.end(),thrust::greater<double>());

    return freq_alpha_lower;
}

int main(int argc, char** argv) {
    if(argc != 3) {
        std::cerr << "Didn't supply plain text and period!" << std::endl;
        return 1;
    }

    std::ifstream ifs(argv[1], std::ios::binary);

    if(!ifs.good()) {
        std::cerr << "Couldn't open text file!" << std::endl;
        return 1;
    }

    unsigned int period = atoi(argv[2]);

    if(period < 4) {
        std::cerr << "Period must be at least 4!" << std::endl;
        return 1;
    }

    // load the file into text
    std::vector<unsigned char> text;

    ifs.seekg(0, std::ios::end); // seek to end of file
    int length = ifs.tellg();    // get distance from beginning
    ifs.seekg(0, std::ios::beg); // move back to beginning

    text.resize(length);
    ifs.read((char*)&text[0], length);

    ifs.close();

    thrust::device_vector<unsigned char> text_clean;
    // TODO: sanitize input to contain only a-z lowercase (use the
    // isnot_lowercase_alpha functor), calculate the number of characters
    // in the cleaned text and put the result in text_clean, make sure to
    // resize text_clean to the correct size!


    int numElements =-1;
    numElements = text.size() - thrust::reduce(
			thrust::make_transform_iterator(text.begin(),isnot_lowercase_alpha()),
			thrust::make_transform_iterator(text.end(),isnot_lowercase_alpha()));

    text_clean.resize(numElements);

    upper_to_lower cast= upper_to_lower();
    thrust::copy_if(thrust::make_transform_iterator(text.begin(),cast),
                    thrust::make_transform_iterator(text.end(),cast),
                    isnot_lowercase_alpha());

    
    std::cout << "\nBefore ciphering!" << std::endl << std::endl;
    std::vector<double> letterFreqGpu = getLetterFrequencyGpu(text_clean);
    std::vector<double> letterFreqCpu = getLetterFrequencyCpu(text);
    bool success = true;
    EXPECT_VECTOR_EQ_EPS(letterFreqCpu, letterFreqGpu, 1e-14, &success);
    PRINT_SUCCESS(success);

    thrust::device_vector<unsigned int> shifts(period);
    // TODO fill in shifts using thrust random number generation (make sure
    // not to allow 0-shifts, this would make for rather poor encryption).


    std::cout << "\nEncryption key: ";

    for(int i = 0; i < period; ++i) {
        std::cout << static_cast<char>('a' + shifts[i]);
    }

    std::cout << std::endl;

    thrust::device_vector<unsigned char> device_cipher_text(numElements);

    // TODO: Apply the shifts to text_clean and place the result in
    // device_cipher_text.

    thrust::host_vector<unsigned char> host_cipher_text = device_cipher_text;

    std::cout << "After ciphering!" << std::endl << std::endl;
    getLetterFrequencyGpu(device_cipher_text);

    std::ofstream ofs("cipher_text.txt", std::ios::binary);

    ofs.write((char*)&host_cipher_text[0], numElements);

    ofs.close();

    return 0;
}
